#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <torch/extension.h>

#define THREADS 256



__global__ void filter_kernel( 
        float *res, 
        float *points,
        const int p_size,
        float* centers,
        int* childIds,
        bool* is_ends
        )
    {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= p_size) return;
    float3 p = reinterpret_cast<float3*> (points + index*3)[0];
    int node_idx = 0;
    while (true){
        if (is_ends[node_idx]){
            reinterpret_cast<float3*>(res + index*3)[0] = make_float3(0.f,0.f,0.f);
            break;
        } 
        int pos = 0;
        float3 center = reinterpret_cast<float3*> (centers + node_idx*3)[0];
        if (p.z < center.z) pos += 4;
        if (p.y < center.y) pos += 2;
        if (p.x < center.x) pos += 1;

        int childId = childIds[node_idx*8 + pos];
        if (childId < 0)
        {reinterpret_cast<float3*>(res + index*3)[0] = p;
        break;}
        node_idx = childId;
    }
        
}

void SVO_filtering( torch::Tensor points, torch::Tensor filtered_points, std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> SVO){
    uint32_t p_size = points.size(0);
    torch::Tensor center_tensor = std::get<0>(SVO);
    torch::Tensor childId_tensor = std::get<1>(SVO);
    torch::Tensor is_end_tensor = std::get<2>(SVO);
    dim3 blocks = (THREADS + p_size)/THREADS;
    filter_kernel<<<blocks, THREADS>>>
    (
     filtered_points.data_ptr<float>(),
     points.data_ptr<float>(), 
     p_size, 
     center_tensor.data_ptr<float>(),
     childId_tensor.data_ptr<int>(),
     is_end_tensor.data_ptr<bool>()
     );

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("run", &SVO_filtering, "build octree from points");
}   